#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

#define SIZE atoi(argv[1])

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		exit(-1);
	}
}

void fill_mat(double *arr, int len)
{
	int i;
	for(i=0;i<len;i++)
		arr[i] = drand48();
}

int main(int argc, char **argv)
{
	if(argc!=2)
	{
		printf("Syntax : exec <size>\n");
		exit(-1);
	}
	
	double *h_A, *h_B;
	double *d_A, *d_B;
	
	hipEvent_t start, stop;

	double time, bandwidth;	
	float diff;

	double time_start, time_end;
        struct timeval tv;
        struct timezone tz;
	
	safe_call(hipEventCreate(&start),__LINE__);
	safe_call(hipEventCreate(&stop),__LINE__);

	h_A = (double *) malloc(SIZE*sizeof(double));
	h_B = (double *) malloc(SIZE*sizeof(double));

	if(h_A==NULL || h_B==NULL)
	{
		printf("Error : host memory allocation\n");
		exit(-1);
	}

	safe_call(hipMalloc((void **)&d_A, SIZE*sizeof(double)),__LINE__);
	safe_call(hipMalloc((void **)&d_B, SIZE*sizeof(double)),__LINE__);

	fill_mat(h_A,SIZE);	

	gettimeofday(&tv, &tz);
        time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
        memcpy((void *)h_B, (void *)h_A, SIZE*sizeof(double)); 
	gettimeofday(&tv, &tz);
        time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
	bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * ( time_end - time_start ) ) ;	
	printf("CPU Memcpy H2H Bandwidth = %f GB/s\n",bandwidth);

	safe_call(hipEventRecord(start, 0),__LINE__);
	safe_call(hipMemcpy((void *)d_A, (void *)h_A, SIZE*sizeof(double), hipMemcpyHostToDevice),__LINE__);
	safe_call(hipEventRecord(stop, 0),__LINE__);
	safe_call(hipEventSynchronize(stop),__LINE__);
	safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
	time = diff*1.0e-3;	
	bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
	printf("CUDA Memcpy H2D Bandwidth = %f GB/s\n",bandwidth);

	safe_call(hipEventRecord(start, 0),__LINE__);
	safe_call(hipMemcpy((void *)d_B, (void *)d_A, SIZE*sizeof(double), hipMemcpyDeviceToDevice),__LINE__);
	safe_call(hipEventRecord(stop, 0),__LINE__);
	safe_call(hipEventSynchronize(stop),__LINE__);
	safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
	time = diff*1.0e-3;	
	bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
	printf("CUDA Memcpy D2D Bandwidth = %f GB/s\n",bandwidth);

	safe_call(hipEventRecord(start, 0),__LINE__);
	safe_call(hipMemcpy((void *)h_B, (void *)d_B, SIZE*sizeof(double), hipMemcpyDeviceToHost),__LINE__);
	safe_call(hipEventRecord(stop, 0),__LINE__);
	safe_call(hipEventSynchronize(stop),__LINE__);
	safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
	time = diff*1.0e-3;	
	bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
	printf("CUDA Memcpy D2H Bandwidth = %f GB/s\n",bandwidth);

	safe_call(hipEventDestroy(start),__LINE__);	
	safe_call(hipEventDestroy(stop),__LINE__);

	safe_call(hipFree(d_A),__LINE__);
	safe_call(hipFree(d_B),__LINE__);
	
	free(h_A);
	free(h_B);
	
	return 0;
}
