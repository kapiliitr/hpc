
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hipblas.h"

void safe_call(hipblasStatus_t ret, int line)
{
	if(ret!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("Error at line %d : %s\n",line);
		exit(-1);
	}
}

int main()
{
	hipblasHandle_t handle;
	int version;
	
	safe_call(hipblasCreate(&handle),__LINE__);
	
	safe_call(cublasGetVersion(handle,&version),__LINE__);

	printf("CUBLAS version = %d\n",version);

	safe_call(hipblasDestroy(handle),__LINE__);
}
