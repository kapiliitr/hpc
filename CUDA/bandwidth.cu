#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<string.h>
#include<assert.h>

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		exit(-1);
	}
}

void fill_mat(double *arr, int len)
{
	int i;
	for(i=0;i<len;i++)
		arr[i] = drand48();
}

int main(int argc, char **argv)
{
	int SIZE, MODE, i; // 0=pageable 1=pinned
	char memmode[10], tempmode[10]; 	

	if(argc<2 || argc>3)
	{
		printf("Syntax : exec -<memory mode> <size>\n");
		exit(-1);
	}
	else if(argc==2)
	{
		MODE = 0;	
		SIZE = atoi(argv[1]);
	}
	else if(argc==3)
	{
		strcpy(tempmode,argv[1]);
		i=0;
		while(tempmode[i]=='-') { i++; }
		if(i==0)
		{
			printf("Syntax : exec -<memory mode> <size>\n");
			exit(-1);
		}
		strcpy(memmode,&tempmode[i]);
		if(strcmp(memmode,"pinned") == 0)
			MODE = 1;
		else if(strcmp(memmode,"pageable") == 0)
			MODE = 0;
		else
		{
			printf("Memory modes pinned and pageable only\n");
			exit(-1);
		}
		SIZE = atoi(argv[2]);
	}
	
	double *h_A, *h_B;
	double *d_A, *d_B;
	
	hipEvent_t start, stop;

	double time, bandwidth;	
	float diff;

	double time_start, time_end;
        struct timeval tv;
        struct timezone tz;
	
	safe_call(hipEventCreate(&start),__LINE__);
	safe_call(hipEventCreate(&stop),__LINE__);
	
	if(MODE==0) //if memory mode = pageable
	{
		h_A = (double *) malloc(SIZE*sizeof(double));
		h_B = (double *) malloc(SIZE*sizeof(double));

		if(h_A==NULL || h_B==NULL)
		{
			printf("Error : host memory allocation\n");
			exit(-1);
		}

		safe_call(hipMalloc((void **)&d_A, SIZE*sizeof(double)),__LINE__);
		safe_call(hipMalloc((void **)&d_B, SIZE*sizeof(double)),__LINE__);

		fill_mat(h_A,SIZE);	
		
		printf("Pageable Memory\n");		

		gettimeofday(&tv, &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		memcpy((void *)h_B, (void *)h_A, SIZE*sizeof(double)); 
		gettimeofday(&tv, &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * ( time_end - time_start ) ) ;	
		printf("CPU Memcpy H2H Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpy((void *)d_A, (void *)h_A, SIZE*sizeof(double), hipMemcpyHostToDevice),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy H2D Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpy((void *)d_B, (void *)d_A, SIZE*sizeof(double), hipMemcpyDeviceToDevice),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy D2D Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpy((void *)h_B, (void *)d_B, SIZE*sizeof(double), hipMemcpyDeviceToHost),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy D2H Bandwidth = %f GB/s\n",bandwidth);
		
		for(i=0;i<SIZE;i++)
			assert(h_A[i]==h_B[i]);

		safe_call(hipFree(d_A),__LINE__);
		safe_call(hipFree(d_B),__LINE__);

		free(h_A);
		free(h_B);
	}
	else //if memory mode = pinned
	{
		safe_call(hipHostMalloc((void **)&h_A, SIZE*sizeof(double), hipHostMallocDefault),__LINE__);
		safe_call(hipHostMalloc((void **)&h_B, SIZE*sizeof(double), hipHostMallocDefault),__LINE__);
	
		safe_call(hipMalloc((void **)&d_A, SIZE*sizeof(double)),__LINE__);
		safe_call(hipMalloc((void **)&d_B, SIZE*sizeof(double)),__LINE__);

		fill_mat(h_A,SIZE);	

		printf("Pinned Memory\n");		

		gettimeofday(&tv, &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		memcpy((void *)h_B, (void *)h_A, SIZE*sizeof(double)); 
		gettimeofday(&tv, &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * ( time_end - time_start ) ) ;	
		printf("CPU Memcpy H2H Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpyAsync((void *)d_A, (void *)h_A, SIZE*sizeof(double), hipMemcpyHostToDevice, 0),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy H2D Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpyAsync((void *)d_B, (void *)d_A, SIZE*sizeof(double), hipMemcpyDeviceToDevice, 0),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy D2D Bandwidth = %f GB/s\n",bandwidth);

		safe_call(hipEventRecord(start, 0),__LINE__);
		safe_call(hipMemcpyAsync((void *)h_B, (void *)d_B, SIZE*sizeof(double), hipMemcpyDeviceToHost, 0),__LINE__);
		safe_call(hipEventRecord(stop, 0),__LINE__);
		safe_call(hipEventSynchronize(stop),__LINE__);
		safe_call(hipEventElapsedTime(&diff,start,stop),__LINE__);
		time = diff*1.0e-3;	
		bandwidth = ( SIZE * sizeof(double) * 2.0 ) / ( 1024 * 1024 * 1024 * time ) ;	
		printf("CUDA Memcpy D2H Bandwidth = %f GB/s\n",bandwidth);

		for(i=0;i<SIZE;i++)
			assert(h_A[i]==h_B[i]);

		safe_call(hipFree(d_A),__LINE__);
		safe_call(hipFree(d_B),__LINE__);
	
		safe_call(hipHostFree(h_A),__LINE__);
		safe_call(hipHostFree(h_B),__LINE__);
	}

	safe_call(hipEventDestroy(start),__LINE__);	
	safe_call(hipEventDestroy(stop),__LINE__);

	return 0;
}
