#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<mpi.h>
#include<string.h>
#include<sys/time.h>

#define SIZE 1024

void safe_call(hipError_t ret, int myrank, int line)
{
	if(ret!=hipSuccess)
	{
		if(myrank == 0)
			printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		MPI_Finalize();
		exit(-1);
	}
}

void fill_data(char *arr, int len)
{
	int i;
	
	for(i=0;i<len;i++)
	{
		srand(time(NULL));
		arr[i] = (char)(rand()%26 + 97);
	}
}

int main(int argc, char *argv[])
{
	int 		comm_size, myrank;
	MPI_Status 	status;	
	char 		myname[MPI_MAX_PROCESSOR_NAME];
	int 		namelen, devcount, device;
	char		*sendbuf, *recvbuf, devname[256];
	int 		i, recvsize, sendsize, *sendcount, *displacement;
	int 		tempdisp;
	hipDeviceProp_t	devprop;
	char 		*h_A, *h_B;
	char 		*d_A, *d_B;
	hipEvent_t 	start, stop;
	double 		time, h2d, d2d, d2h;	
	float 		diff;

	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&comm_size);
	MPI_Comm_rank(MPI_COMM_WORLD,&myrank);

	MPI_Get_processor_name(myname, &namelen);
	myname[namelen++] = (char)0;         
	
	safe_call(hipGetDeviceCount(&devcount),myrank,__LINE__);		

	if(myrank == 0)
	{
		sendsize = devcount*SIZE*sizeof(char);
		sendbuf = (char *) malloc(sendsize);
		fill_data(sendbuf,devcount*SIZE);
	}
	
	sendcount = (int *) malloc(comm_size*sizeof(int));
	displacement = (int *) malloc(comm_size*sizeof(int));
	
	tempdisp = 0;
	for(i = 0; i < ((comm_size<devcount)?comm_size:devcount); i++)
	{	
		displacement[i] = tempdisp;

		if(devcount%comm_size == 0)
			sendcount[i] = (devcount/comm_size)*SIZE;
		else
		{
			if(myrank < (devcount % comm_size))
				sendcount[i] = (devcount/comm_size + 1)*SIZE;
			else	
				sendcount[i] = (devcount/comm_size)*SIZE;
		}

		tempdisp += sendcount[i];
	}

	if(devcount % comm_size == 0)	
		recvsize = (devcount/comm_size)*SIZE;
	else
	{
		if(myrank < (devcount % comm_size))
			recvsize = (devcount/comm_size + 1)*SIZE;
		else	
			recvsize= (devcount/comm_size)*SIZE;
	}
	recvbuf = (char *) malloc(recvsize*sizeof(char));		
	
	MPI_Barrier(MPI_COMM_WORLD);
	
	MPI_Scatterv(&sendbuf,sendcount,displacement,MPI_CHAR,&recvbuf,recvsize,MPI_CHAR,0,MPI_COMM_WORLD);
	
	MPI_Barrier(MPI_COMM_WORLD);

	safe_call(hipEventCreate(&start),myrank,__LINE__);
	safe_call(hipEventCreate(&stop),myrank,__LINE__);

	for(i = myrank; i < devcount; i+=comm_size)
	{
		safe_call(hipSetDevice(i),myrank,__LINE__);		
		safe_call(hipGetDevice(&device),myrank,__LINE__);
		
		if(device == i)
		{
			safe_call(hipGetDeviceProperties(&devprop,device),myrank,__LINE__);
			strcpy(devname,devprop.name);		
			
			h_A = (char *) malloc(SIZE*sizeof(char));	
			h_B = (char *) malloc(SIZE*sizeof(char));		

			if(h_A==NULL || h_B==NULL)
			{
				if(myrank == 0)
					printf("Error : host memory allocation, Line : %d\n",myrank,__LINE__);
				MPI_Finalize();
				exit(-1);
			}
	
			safe_call(hipMalloc((void **)&d_A, SIZE*sizeof(char)),myrank,__LINE__);
			safe_call(hipMalloc((void **)&d_B, SIZE*sizeof(char)),myrank,__LINE__);

			memcpy(h_A,&recvbuf[((i-myrank)/comm_size)*SIZE],SIZE*sizeof(char));			
			
			/************************************** Host to Device Starts ***********************************/
			safe_call(hipEventRecord(start, 0),myrank,__LINE__);
			
			safe_call(hipMemcpy((void *)d_A, (void *)h_A, SIZE*sizeof(char), hipMemcpyHostToDevice),myrank,__LINE__);
			
			safe_call(hipEventRecord(stop, 0),myrank,__LINE__);
			safe_call(hipEventSynchronize(stop),myrank,__LINE__);
			
			safe_call(hipEventElapsedTime(&diff,start,stop),myrank,__LINE__);
			
			time = diff*1.0e-3;	
			h2d = ( SIZE * sizeof(char) * 2.0 ) / ( 1024 * 1024 * time ) ;	
			/************************************** Host to Device Ends **************************************/
	
			/************************************** Device to Device Starts **********************************/
			safe_call(hipEventRecord(start, 0),myrank,__LINE__);
			
			safe_call(hipMemcpy((void *)d_B, (void *)d_A, SIZE*sizeof(char), hipMemcpyDeviceToDevice),myrank,__LINE__);
			
			safe_call(hipEventRecord(stop, 0),myrank,__LINE__);
			safe_call(hipEventSynchronize(stop),myrank,__LINE__);
			
			safe_call(hipEventElapsedTime(&diff,start,stop),myrank,__LINE__);
			
			time = diff*1.0e-3;	
			d2d = ( SIZE * sizeof(char) * 2.0 ) / ( 1024 * 1024 * time ) ;	
			/************************************** Device to Device Ends ************************************/

			/************************************** Device to Host Starts ************************************/	
			safe_call(hipEventRecord(start, 0),myrank,__LINE__);
			
			safe_call(hipMemcpy((void *)h_B, (void *)d_B, SIZE*sizeof(char), hipMemcpyDeviceToHost),myrank,__LINE__);
			
			safe_call(hipEventRecord(stop, 0),myrank,__LINE__);
			safe_call(hipEventSynchronize(stop),myrank,__LINE__);
			
			safe_call(hipEventElapsedTime(&diff,start,stop),myrank,__LINE__);
			
			time = diff*1.0e-3;	
			d2h = ( SIZE * sizeof(char) * 2.0 ) / ( 1024 * 1024 * time ) ;	
			/************************************** Device to Host Ends **************************************/	

			printf("Device %d : %s\n \
				Host to Device : %fMB/s\n \
				Device to Device : %fMB/s\n \
				Device to Host : %fMB/s\n \
				",device,devname,h2d,d2d,d2h);
			
			safe_call(hipFree(d_A),myrank,__LINE__);
			safe_call(hipFree(d_B),myrank,__LINE__);

			free(h_A);
			free(h_B);
		}
	}
	
	safe_call(hipEventDestroy(start),myrank,__LINE__);	
	safe_call(hipEventDestroy(stop),myrank,__LINE__);

	MPI_Finalize();
	
	return 0;
}
